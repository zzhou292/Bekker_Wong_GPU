#include "hip/hip_runtime.h"
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>
#include <fstream>
#include <iostream>
#include <string>
#include <boost/filesystem.hpp>
#include <vector>
#include <cmath>
#include "BWTerrain.cuh"

BWTerrain::BWTerrain(float x_in, float y_in, float resolution_in) {
    x = x_in;
    y = y_in;
    resolution = resolution_in;
}

void BWTerrain::Initialize() {
    x_n_node = x / resolution + 1;
    y_n_node = y / resolution + 1;
    n_node = x_n_node * y_n_node;

    x_arr = new float[n_node];
    y_arr = new float[n_node];
    z_arr = new float[n_node];

    for (int i = 0; i < n_node; i++) {
        x_arr[i] = (i % x_n_node) * resolution;
        y_arr[i] = (int)(i / x_n_node) * resolution;
        z_arr[i] = 0.f;
    }

    // malloc GPU memory
    hipMalloc((float**)&gpu_x_arr, n_node * sizeof(float));
    hipMalloc((float**)&gpu_y_arr, n_node * sizeof(float));
    hipMalloc((float**)&gpu_z_arr, n_node * sizeof(float));

    // copy CPU data to GPU
    hipMemcpy(gpu_x_arr, x_arr, n_node * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_y_arr, y_arr, n_node * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_z_arr, z_arr, n_node * sizeof(float), hipMemcpyHostToDevice);
}

void BWTerrain::Destroy() {
    hipFree(gpu_x_arr);
    hipFree(gpu_y_arr);
    hipFree(gpu_z_arr);
}

void BWTerrain::Advance(float time_step, BWWheel* wheel) {
    float x_min = wheel->pos_x - wheel->Get_R();
    float x_max = wheel->pos_x + wheel->Get_R();

    float y_min = wheel->pos_y - wheel->Get_W() / 2.f;
    float y_max = wheel->pos_y + wheel->Get_W() / 2.f;

    // find all vertices in the region of the cylinder
    std::vector<int> active_idx = Util_Find_Active(x_min, x_max, y_min, y_max);

    std::cout << "num_active: " << active_idx.size() << std::endl;
    int* active_arr = active_idx.data();
    int size = active_idx.size();
    // std::cout << "active:" << size << stUtil_Find_Activendl;
    BWTerrain::Util_Compute_Internal_Force(active_arr, size, wheel);

    hipMemcpy(x_arr, gpu_x_arr, n_node * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y_arr, gpu_y_arr, n_node * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(z_arr, gpu_z_arr, n_node * sizeof(float), hipMemcpyDeviceToHost);
}

void BWTerrain::WriteOutput(std::string FileName) {
    boost::filesystem::path dir("OUTPUT");

    if (!(boost::filesystem::exists(dir))) {
        std::cout << " Output Folder Doesn't Exists" << std::endl;

        if (boost::filesystem::create_directory(dir))
            std::cout << "....Successfully Created !" << std::endl;
    }

    // Create and open a text file
    std::ofstream OutOBJ("OUTPUT/" + FileName + ".obj");

    for (int i = 0; i < n_node; i++) {
        OutOBJ << "v"
               << " " << x_arr[i] << " " << y_arr[i] << " " << z_arr[i] << std::endl;
    }

    for (int j = 0; j < y_n_node - 1; j++) {
        for (int i = 0; i < x_n_node - 1; i++) {
            OutOBJ << "f"
                   << " " << j * x_n_node + i + 1 << " " << j * x_n_node + i + 1 + 1 << " "
                   << (j + 1) * x_n_node + i + 1 << std::endl;
            OutOBJ << "f"
                   << " " << j * x_n_node + i + 1 + 1 << " " << (j + 1) * x_n_node + i + 1 << " "
                   << (j + 1) * x_n_node + i + 1 + 1 << std::endl;
        }
    }

    // Close the file
    OutOBJ.close();
}

// Utility Funtions:
__global__ void Ker_Find_Active(float* gpu_x_in,
                                float* gpu_y_in,
                                float x_min,
                                float x_max,
                                float y_min,
                                float y_max,
                                int size,
                                bool* out_bool) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx > size)
        return;
    if (gpu_x_in[idx] >= x_min && gpu_x_in[idx] <= x_max && gpu_y_in[idx] >= y_min && gpu_y_in[idx] <= y_max)
        out_bool[idx] = true;
    __syncthreads();
}

// Note: This util function assumes that the out_idx array is a float array uninitialized
std::vector<int> BWTerrain::Util_Find_Active(float x_min, float x_max, float y_min, float y_max) {
    int block_size = 1024;
    int n_block = n_node / 1024 + 1;

    bool* out_bool = new bool[n_node];
    for (int i = 0; i < n_node; i++) {
        out_bool[i] = false;
    }
    bool* gpu_out_bool;
    hipMalloc((bool**)&gpu_out_bool, n_node * sizeof(bool));
    hipMemcpy(gpu_out_bool, out_bool, n_node * sizeof(bool), hipMemcpyHostToDevice);

    Ker_Find_Active<<<n_block, block_size>>>(gpu_x_arr, gpu_y_arr, x_min, x_max, y_min, y_max, n_node, gpu_out_bool);

    hipMemcpy(out_bool, gpu_out_bool, n_node * sizeof(bool), hipMemcpyDeviceToHost);

    std::vector<int> idx_vec;

    for (int i = 0; i < n_node; i++) {
        if (out_bool[i] == true) {
            idx_vec.push_back(i);
        }
    }

    hipFree(gpu_out_bool);

    return idx_vec;
}

// Utility Funtions:
__global__ void Ker_Compute_Force(float* gpu_x_in,
                                  float* gpu_y_in,
                                  float* gpu_z_in,
                                  int* active_idx,
                                  int idx_size,
                                  float pos_x,
                                  float pos_y,
                                  float pos_z,
                                  float r,
                                  float* gpu_out_force) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx > idx_size)
        return;

    // Current vertex z direction ray-casting
    float a = abs(gpu_x_in[active_idx[idx]]) - pos_x;
    float c = sqrt(pow(a, 2) + pow(r, 2));
    float wheel_z = r * (c - r) / c;
    float delta_z = gpu_z_in[active_idx[idx]] - wheel_z;

    gpu_z_in[active_idx[idx]] = wheel_z;

    // generate a fictitious force
    if (delta_z > 0)
        gpu_out_force[active_idx[idx]] = delta_z * 1000.f;

    __syncthreads();
}

// Utility function for z-direction ray casting and internal force computation
void BWTerrain::Util_Compute_Internal_Force(int* idx_arr, int idx_arr_size, BWWheel* wheel) {
    float* out_force = new float[n_node];
    float* gpu_out_force;

    int block_size = 1024;
    int n_block = idx_arr_size / 1024 + 1;

    for (int i = 0; i < n_node; i++) {
        out_force[i] = 0.f;
    }

    hipMalloc((float**)&gpu_out_force, n_node * sizeof(float));

    hipMemcpy(gpu_out_force, out_force, n_node * sizeof(float), hipMemcpyHostToDevice);

    Ker_Compute_Force<<<n_block, block_size>>>(gpu_x_arr, gpu_y_arr, gpu_z_arr, idx_arr, idx_arr_size, wheel->pos_x,
                                               wheel->pos_y, wheel->pos_z, wheel->Get_R(), gpu_out_force);

    hipMemcpy(out_force, gpu_out_force, n_node * sizeof(float), hipMemcpyDeviceToHost);

    float sum_force = 0.f;
    for (int i = 0; i < n_node; i++) {
        sum_force += out_force[i];
    }
    std::cout << "sum_force:" << sum_force << std::endl;
    wheel->acc_z = sum_force / wheel->Get_M() - 9.8f;

    hipFree(gpu_out_force);
}