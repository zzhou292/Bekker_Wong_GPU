#include "hip/hip_runtime.h"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <fstream>
#include <iostream>
#include <string>
#include <algorithm>
#include <boost/filesystem.hpp>
#include <vector>
#include <cmath>
#include "BWTerrain.cuh"

BWTerrain::BWTerrain(float x_in, float y_in, float resolution_in) {
    x = x_in;
    y = y_in;
    resolution = resolution_in;
}

void BWTerrain::Initialize() {
    // Initialize terrain size and area information
    x_n_node = x / resolution + 1;
    y_n_node = y / resolution + 1;
    n_node = x_n_node * y_n_node;
    area = resolution * resolution;

    x_arr = new float[n_node];
    y_arr = new float[n_node];
    z_arr = new float[n_node];

    for (int i = 0; i < n_node; i++) {
        x_arr[i] = (i % x_n_node) * resolution;
        y_arr[i] = (int)(i / x_n_node) * resolution;
        z_arr[i] = 0.f;
    }

    // initialize unified memory for Bekker soil parameter structure
    // set all parameters to default values
    hipMallocManaged(&terrain_params, sizeof(BWParameters));
    terrain_params->Kphi = 0.2e6;
    terrain_params->Kc = 0;
    terrain_params->n = 1.1;
    terrain_params->f_s = 0.0;
    terrain_params->bz_ratio = 0.2;

    // malloc GPU memory for z array
    hipMalloc((float**)&gpu_x_arr, n_node * sizeof(float));
    hipMalloc((float**)&gpu_y_arr, n_node * sizeof(float));
    hipMalloc((float**)&gpu_z_arr, n_node * sizeof(float));

    // copy CPU data to GPU
    hipMemcpy(gpu_x_arr, x_arr, n_node * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_y_arr, y_arr, n_node * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_z_arr, z_arr, n_node * sizeof(float), hipMemcpyHostToDevice);
}

void BWTerrain::Destroy() {
    // detroy all allocated GPU memory
    hipFree(gpu_x_arr);
    hipFree(gpu_y_arr);
    hipFree(gpu_z_arr);
    hipFree(terrain_params);
}

void BWTerrain::Advance(float time_step, BWWheel* wheel) {
    // calculate x boundary
    float x_min = wheel->pos_x - wheel->Get_R();
    float x_max = wheel->pos_x + wheel->Get_R();

    // calculate y boundary
    float y_min = wheel->pos_y - wheel->Get_W() / 2.f;
    float y_max = wheel->pos_y + wheel->Get_W() / 2.f;

    // find all vertices in the region of the cylinder - we call them active vertices
    std::vector<int> active_idx = Util_Find_Active(x_min, x_max, y_min, y_max);

    // convert std::vector into int array for CUDA
    int* active_arr = active_idx.data();
    int size = active_idx.size();
    if (enable_bulldozing)
        displacement_arr = new float[size];

    // compute internal force
    BWTerrain::Util_Compute_Internal_Force(active_arr, size, wheel, displacement_arr);

    // copy updated z array back to CPU
    hipMemcpy(z_arr, gpu_z_arr, n_node * sizeof(float), hipMemcpyDeviceToHost);

    // compute bulldozing force
    if (enable_bulldozing) {
        BWTerrain::Util_Compute_Bulldozing(active_arr, displacement_arr, size);
    }

    // copy updated z array back to CPU
    hipMemcpy(z_arr, gpu_z_arr, n_node * sizeof(float), hipMemcpyDeviceToHost);
}

void BWTerrain::SetBWParams(BWParameters* params_in) {
    // update all terrain parameters based on the BWParameters input
    terrain_params->Kphi = params_in->Kphi;
    terrain_params->Kc = params_in->Kc;
    terrain_params->n = params_in->n;
    terrain_params->f_s = params_in->f_s;
    terrain_params->bz_ratio = params_in->bz_ratio;
}

void BWTerrain::WriteOutput(std::string FileName) {
    boost::filesystem::path dir("OUTPUT");

    if (!(boost::filesystem::exists(dir))) {
        std::cout << " Output Folder Doesn't Exists" << std::endl;

        if (boost::filesystem::create_directory(dir))
            std::cout << "....Successfully Created !" << std::endl;
    }

    // create and open an obj file
    std::ofstream OutOBJ("OUTPUT/" + FileName + ".obj");

    for (int i = 0; i < n_node; i++) {
        OutOBJ << "v"
               << " " << x_arr[i] << " " << y_arr[i] << " " << z_arr[i] << std::endl;
    }

    // write out all vertices and faces information
    for (int j = 0; j < y_n_node - 1; j++) {
        for (int i = 0; i < x_n_node - 1; i++) {
            OutOBJ << "f"
                   << " " << j * x_n_node + i + 1 << " " << j * x_n_node + i + 1 + 1 << " "
                   << (j + 1) * x_n_node + i + 1 << std::endl;
            OutOBJ << "f"
                   << " " << j * x_n_node + i + 1 + 1 << " " << (j + 1) * x_n_node + i + 1 << " "
                   << (j + 1) * x_n_node + i + 1 + 1 << std::endl;
        }
    }

    // close the file
    OutOBJ.close();
}

// CUDA kernel call to find active vertices
__global__ void Ker_Find_Active(float* gpu_x_in,
                                float* gpu_y_in,
                                float x_min,
                                float x_max,
                                float y_min,
                                float y_max,
                                int size,
                                bool* out_bool) {
    // find the id for the current thread
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // if out of range, quick rejection
    if (idx >= size)
        return;

    // if inside interest region, update out_bool arr
    if (gpu_x_in[idx] >= x_min && gpu_x_in[idx] <= x_max && gpu_y_in[idx] >= y_min && gpu_y_in[idx] <= y_max)
        out_bool[idx] = true;

    // waiting for all threads to finish
    __syncthreads();
}

// Wrapper for CUDA kernal call to find all active vertices
std::vector<int> BWTerrain::Util_Find_Active(float x_min, float x_max, float y_min, float y_max) {
    // default block size set to 1024
    int block_size = 1024;
    int n_block = n_node / block_size + 1;

    // create and copy an output boolean array into GPU memory
    bool* out_bool = new bool[n_node];
    for (int i = 0; i < n_node; i++) {
        out_bool[i] = false;
    }
    bool* gpu_out_bool;
    hipMalloc((bool**)&gpu_out_bool, n_node * sizeof(bool));
    hipMemcpy(gpu_out_bool, out_bool, n_node * sizeof(bool), hipMemcpyHostToDevice);

    // call CUDA kernel
    Ker_Find_Active<<<n_block, block_size>>>(gpu_x_arr, gpu_y_arr, x_min, x_max, y_min, y_max, n_node, gpu_out_bool);

    // copy data back to cpu array
    hipMemcpy(out_bool, gpu_out_bool, n_node * sizeof(bool), hipMemcpyDeviceToHost);

    // store all active vertices in a std::vector
    std::vector<int> idx_vec;
    for (int i = 0; i < n_node; i++) {
        if (out_bool[i] == true) {
            idx_vec.push_back(i);
        }
    }

    // free temporary GPU memory
    hipFree(gpu_out_bool);

    return idx_vec;
}

// CUDA kernel call to compute force based on Bekker-Wong Pressure-Sinkage Formulation
__global__ void Ker_Compute_Force(float* gpu_x_in,
                                  float* gpu_y_in,
                                  float* gpu_z_in,
                                  int* active_idx,
                                  float* gpu_displacement_arr,
                                  int idx_size,
                                  float pos_x,
                                  float pos_y,
                                  float pos_z,
                                  float r,
                                  float area,
                                  float b,
                                  float* gpu_out_force,
                                  BWParameters* params_in,
                                  bool bulldozing) {
    // calculate the idx for the current thread
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // if out of range, return
    if (idx >= idx_size)
        return;

    // current vertex z direction ray-casting
    float a = fabsf(gpu_x_in[active_idx[idx]]) - pos_x;
    float absz = sqrtf(powf(r, 2) - powf(a, 2));
    // ray-casting: the lowest z direction on the cylinderical wheel
    float wheel_z = pos_z - absz;
    float delta_z = gpu_z_in[active_idx[idx]] - wheel_z;
    // generate force based on BW formula
    if (delta_z > 0) {
        float p_pressure = (params_in->Kc / b + params_in->Kphi) * powf(delta_z, params_in->n);
        gpu_out_force[active_idx[idx]] = p_pressure * area;
        gpu_z_in[active_idx[idx]] = wheel_z;
        if (bulldozing)
            gpu_displacement_arr[idx] = delta_z;
    }

    // waiting for all threads to finish
    __syncthreads();
}

// Wrapper for CUDA kernal call to compute force based on Bekker-Wong Formulation
void BWTerrain::Util_Compute_Internal_Force(int* idx_arr, int idx_arr_size, BWWheel* wheel, float* displacement_arr) {
    float* out_force = new float[n_node];
    int* gpu_idx_arr;
    float* gpu_out_force;

    float* gpu_displacement_arr;

    int block_size = 1024;
    int n_block = idx_arr_size / 1024 + 1;

    for (int i = 0; i < n_node; i++) {
        out_force[i] = 0.f;
    }

    hipMalloc((float**)&gpu_out_force, n_node * sizeof(float));
    hipMalloc((int**)&gpu_idx_arr, idx_arr_size * sizeof(int));
    if (enable_bulldozing)
        hipMalloc((float**)&gpu_displacement_arr, idx_arr_size * sizeof(float));

    hipMemcpy(gpu_out_force, out_force, n_node * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_idx_arr, idx_arr, idx_arr_size * sizeof(int), hipMemcpyHostToDevice);

    float b = resolution * resolution / (resolution * 4);

    // call CUDA kernal
    Ker_Compute_Force<<<n_block, block_size>>>(gpu_x_arr, gpu_y_arr, gpu_z_arr, gpu_idx_arr, gpu_displacement_arr,
                                               idx_arr_size, wheel->pos_x, wheel->pos_y, wheel->pos_z, wheel->Get_R(),
                                               area, b, gpu_out_force, terrain_params, enable_bulldozing);

    hipMemcpy(out_force, gpu_out_force, n_node * sizeof(float), hipMemcpyDeviceToHost);
    if (enable_bulldozing)
        hipMemcpy(displacement_arr, gpu_displacement_arr, idx_arr_size * sizeof(float), hipMemcpyDeviceToHost);

    float sum_force = 0.f;
    for (int i = 0; i < n_node; i++) {
        sum_force += out_force[i];
    }

    wheel->acc_z = sum_force / wheel->Get_M() - 9.8f;

    hipFree(gpu_out_force);
    hipFree(gpu_idx_arr);
    if (enable_bulldozing)
        hipFree(gpu_displacement_arr);
}

__global__ void Ker_Get_Bz_Neighbours(int* idx_in, int* idx_out, int size, int x_node_num, int y_node_num) {
    // calculate the idx for the current thread
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // if out of range, return
    if (idx >= size)
        return;

    int y_idx = (int)idx_in[idx] / x_node_num;
    int x_idx = idx_in[idx] % x_node_num;

    int x_idx_1 = x_idx + 1;
    int x_idx_2 = x_idx - 1;
    int y_idx_1 = y_idx + 1;
    int y_idx_2 = y_idx - 1;

    if (x_idx_1 >= x_node_num) {
        idx_out[4 * idx] = -1;
    } else {
        // idx_out[4 * idx] = y_idx * x_node_num + x_idx_1;
        idx_out[4 * idx] = idx_in[idx] + 1;
    }

    if (x_idx_2 < 0) {
        idx_out[4 * idx + 1] = -1;
    } else {
        // idx_out[4 * idx + 1] = y_idx * x_node_num + x_idx_2;
        idx_out[4 * idx + 1] = idx_in[idx] - 1;
    }

    if (y_idx_1 >= y_node_num) {
        idx_out[4 * idx + 2] = -1;
    } else {
        // idx_out[4 * idx + 2] = y_idx_1 * x_node_num + x_idx;
        idx_out[4 * idx + 2] = idx_in[idx] + x_node_num;
    }

    if (y_idx_2 < 0) {
        idx_out[4 * idx + 3] = -1;
    } else {
        // idx_out[4 * idx + 3] = y_idx_2 * x_node_num + x_idx;
        idx_out[4 * idx + 3] = idx_in[idx] - x_node_num;
    }

    __syncthreads();
}

__global__ void Ker_Bz_Raise_Neighbour(int* idx_in, int size, float* gpu_z_arr, float avg_soil_raise, float bz_ratio) {
    // calculate the idx for the current thread
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // if out of range, return
    if (idx >= size)
        return;

    gpu_z_arr[idx_in[idx]] += bz_ratio * avg_soil_raise;
}

void BWTerrain::Util_Compute_Bulldozing(int* idx_in, float* displacement_in, int active_size) {
    float tot_displacement = 0.f;
    std::vector<int> hit_vertices;
    for (int i = 0; i < active_size; i++) {
        if (displacement_in[i] > 0) {
            hit_vertices.push_back(idx_in[i]);
            tot_displacement += displacement_in[i];
        }
    }

    int hit_size = hit_vertices.size();

    if (hit_size == 0) {
        return;
    }

    int* hit_arr = hit_vertices.data();

    int* gpu_idx_in;   // input index array, including all hit vertices from the last loop
    int* gpu_idx_out;  // output index array containing the neighbour of all 'hit_size' vertices
    int* neighbour_idx = new int[4 * hit_size];

    hipMalloc((int**)&gpu_idx_in, hit_size * sizeof(int));
    hipMalloc((int**)&gpu_idx_out, 4 * hit_size * sizeof(int));

    hipMemcpy(gpu_idx_in, hit_arr, hit_size * sizeof(int), hipMemcpyHostToDevice);

    int block_size = 1024;
    int n_block = hit_size / 1024 + 1;

    Ker_Get_Bz_Neighbours<<<n_block, block_size>>>(gpu_idx_in, gpu_idx_out, hit_size, x_n_node, y_n_node);

    hipMemcpy(neighbour_idx, gpu_idx_out, 4 * hit_size * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<int> raw_neighbour;
    for (int i = 0; i < 4 * hit_size; i++) {
        raw_neighbour.push_back(neighbour_idx[i]);
    }

    std::sort(raw_neighbour.begin(), raw_neighbour.end());

    std::vector<int>::iterator ip;
    ip = std::unique(raw_neighbour.begin(), raw_neighbour.begin() + raw_neighbour.size());
    raw_neighbour.resize(std::distance(raw_neighbour.begin(), ip));

    if (raw_neighbour[0] == -1) {
        raw_neighbour.erase(raw_neighbour.begin() + 0);
    }

    hipFree(gpu_idx_in);
    hipFree(gpu_idx_out);

    int neigh_size = raw_neighbour.size();
    int* neigh_arr = raw_neighbour.data();

    int* gpu_neigh_arr;

    hipMalloc((int**)&gpu_neigh_arr, neigh_size * sizeof(int));
    hipMemcpy(gpu_neigh_arr, neigh_arr, neigh_size * sizeof(int), hipMemcpyHostToDevice);

    float avg_soil_raise = tot_displacement / neigh_size;

    // Raise Neighourbing Nodes
    Ker_Bz_Raise_Neighbour<<<n_block, block_size>>>(gpu_neigh_arr, neigh_size, gpu_z_arr, avg_soil_raise,
                                                    terrain_params->bz_ratio);

    hipFree(gpu_neigh_arr);
}
